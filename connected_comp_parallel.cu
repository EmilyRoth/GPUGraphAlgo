
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>

using namespace std;

__global__ void befriend_adjacents(int* adj_lists, int* sizes, int* labels, int* changed) {
    int id = threadIdx.x;

    // Get start of adj list
    int begin = 0;
    for (int i = 0; i < id; i++) {
        begin += sizes[i];
    }

    int* list = &adj_lists[begin];

    // debug
    // if (sizes[id] == 0) {
    //     debug[id] = -1;
    // } else {
    //     debug[id] = list[0];
    // }

    int min = labels[id];
    for (int i = 0; i < sizes[id]; i++) {
        int adj_label = labels[list[i]];
        if (adj_label < min) {
            min = adj_label;
            *changed = 1;
        }
    }
    labels[id] = min;
}

__global__ void init_labels(int* labels) {
    int id = threadIdx.x;
    labels[id] = id;
}

void print_labels(int* labels, int size) {
    cout << "labels: ";
    for (int i = 0; i < size; i++) {
        cout << labels[i] << " ";
    }
    cout << endl;
}

void print_lists(int* adj_lists, int size, int* list_sizes) {
    cout << "adjacency lists:" << endl;
    int* list = adj_lists;
    for (int i = 0; i < size; i++) {
        cout << "[";
        for (int j = 0; j < list_sizes[i]; j++) {
            cout << " " << list[j] << " ";
        }
        cout << "]" << endl;
        list += list_sizes[i];
    }
}
// void print_debug(int* debug, int size) {
//     cout << "debug: ";
//     for (int i = 0; i < size; i++) {
//         cout << debug[i] << " ";
//     }
//     cout << endl;
// }

// adj_lists is flattened to 1d array
void cc_para(int* adj_lists, int size, int* list_sizes) {
    int* labels = (int*)malloc(size * sizeof(int));
    int* d_labels;

    int list_size_total = 0;
    for (int i = 0; i < size; i++) {
        list_size_total += list_sizes[i];
    }

    hipMalloc((void**)&d_labels, size * sizeof(int));

    init_labels<<<1, size>>>(d_labels);

    hipMemcpy(labels, d_labels, size * sizeof(int), hipMemcpyDeviceToHost);

    int* d_adj_lists;
    hipMalloc((void**)&d_adj_lists, list_size_total * sizeof(int));
    hipMemcpy(d_adj_lists, adj_lists, list_size_total * sizeof(int), hipMemcpyHostToDevice);

    int* d_sizes;
    hipMalloc((void**)&d_sizes, size * sizeof(int));
    hipMemcpy(d_sizes, list_sizes, size * sizeof(int), hipMemcpyHostToDevice);

    int* changed = (int*)malloc(sizeof(int));
    int* d_changed;
    hipMalloc((void**)&d_changed, sizeof(int));
    *changed = 1;

    while (*changed != 0) {
        *changed = 0;
        hipMemcpy(d_changed, changed, sizeof(int), hipMemcpyHostToDevice);

        befriend_adjacents<<<1, size>>>(d_adj_lists, d_sizes, d_labels, d_changed);

        hipMemcpy(changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
    }

    hipMemcpy(labels, d_labels, size * sizeof(int), hipMemcpyDeviceToHost);

    print_labels(labels, size);

    hipFree(d_changed); hipFree(d_labels); hipFree(d_changed); hipFree(d_adj_lists);

    free(labels); free(changed);
}

int main() {
    // int * adj_lists[3];
    // int* list0;
    // int list1[1] = {2};
    // int list2[2] = {1}; 
    // adj_lists[0] = list0;
    // adj_lists[1] = list1;
    // adj_lists[2] = list2;
    int adj_lists[4] = {1, 2, 0, 0};

    int size = 4;
    int sizes[4] = {2, 1, 1, 0};

    print_lists(adj_lists, size, sizes);

    cc_para(adj_lists, size, sizes);
}