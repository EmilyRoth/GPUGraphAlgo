#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>

__global__ void befriend_adjacents(int** adj_lists, int* sizes, int* labels, int* changed) {
    int id = threadIdx.x;
    int* list = adj_lists[id];
    int min = labels[id];
    for (int i = 0; i < sizes[id]; i++) {
        int adj_label = labels[list[i]]
        if (adj_label < min) {
            min = adj_label;
            *changed = 1;
        }
    }
    labels[id] = min;
}

__global__ void init_labels(int* labels) {
    int id = threadIdx.x;
    label[id] = id;
}

void cc_para(int** adj_lists, int size, int* list_sizes) {
    int* labels = (int*)malloc(size * sizeof(int));
    int* d_labels;
    hipMalloc((void**)&d_labels, size * sizeof(int));

    init_labels<<<1, size>>>(d_labels)

    int** d_adj_lists;
    hipMalloc((void**)&d_adj_lists, size * sizeof(int*));
    for (int i = 0; i < size; i++) {
        hipMalloc((void**)d_adj_lists[i], list_sizes[i] * sizeof(int));
        hipMemcpy(*(&d_adj_lists[i]), &adj_lists[i], list_sizes[i] * sizeof(int), hipMemcpyHostToDevice);
    }
    int* d_sizes;
    hipMalloc((void**)&d_sizes, size * sizeof(int));
    hipMemcpy(d_sizes, sizes, size * sizeof(int), hipMemcpyHostToDevice);

    int* changed = (int*)malloc(sizeof(int));
    int* d_changed;
    hipMalloc((void**)&d_changed, sizeof(int));
    *changed = 1;

    while (*changed != 0) {
        *changed = 0;
        hipMemcpy(d_changed, changed, sizeof(int), hipMemcpyHostToDevice);

        befriend_adjacents<<<1, size>>>(d_adj_lists, d_sizes, d_labels, d_changed);

        hipMemcpy(changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
    }

    hipMemcpy(labels, d_labels, size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        hipFree(d_adj_lists[i]);
    }
    hipFree(d_changed); hipFree(d_labels); hipFree(d_changed); hipFree(d_adj_lists);

    free(labels); free(changed);
}

int main() {

}