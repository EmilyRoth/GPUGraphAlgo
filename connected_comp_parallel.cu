
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include "vector"
#include <sstream>
#include <string>
using namespace std;


using namespace std;

__global__ void befriend_adjacents(int* adj_lists, int* sizes, int* labels, int* changed) {
    int id = threadIdx.x;

    // Get start of adj list
    int begin = 0;
    for (int i = 0; i < id; i++) {
        begin += sizes[i];
    }

    int* list = &adj_lists[begin];

    // debug
    // if (sizes[id] == 0) {
    //     debug[id] = -1;
    // } else {
    //     debug[id] = list[0];
    // }

    int min = labels[id];
    for (int i = 0; i < sizes[id]; i++) {
        int adj_label = labels[list[i]];
        if (adj_label < min) {
            min = adj_label;
            *changed = 1;
        }
    }
    labels[id] = min;
}

__global__ void init_labels(int* labels) {
    int id = threadIdx.x;
    labels[id] = id;
}

void print_labels(int* labels, int size) {
		ofstream myfile ("cuda_output.group");
    cout << "labels: ";
    for (int i = 0; i < size; i++) {
        cout << labels[i] << " ";
        myfile << labels[i] << " ";
    }
    cout << endl;
		myfile << endl;
}

void print_lists(int* adj_lists, int size, int* list_sizes) {
		ofstream myfile ("cuda_output.adjlist");
    cout << "adjacency lists:" << endl;
    int* list = adj_lists;
    for (int i = 0; i < size; i++) {
        cout << "[";
				myfile << i << " ";
        for (int j = 0; j < list_sizes[i]; j++) {
            cout << " " << list[j] << " ";
						myfile << list[j]  << " ";
        }
        cout << "]" << endl;
				myfile << endl;
        list += list_sizes[i];
    }
		myfile.close();	
}
// void print_debug(int* debug, int size) {
//     cout << "debug: ";
//     for (int i = 0; i < size; i++) {
//         cout << debug[i] << " ";
//     }
//     cout << endl;
// }

// adj_lists is flattened to 1d array
void cc_para(int* adj_lists, int size, int* list_sizes) {
    int* labels = (int*)malloc(size * sizeof(int));
    int* d_labels;

    int list_size_total = 0;
    for (int i = 0; i < size; i++) {
        list_size_total += list_sizes[i];
    }

    hipMalloc((void**)&d_labels, size * sizeof(int));

    init_labels<<<1, size>>>(d_labels);

    hipMemcpy(labels, d_labels, size * sizeof(int), hipMemcpyDeviceToHost);

    int* d_adj_lists;
    hipMalloc((void**)&d_adj_lists, list_size_total * sizeof(int));
    hipMemcpy(d_adj_lists, adj_lists, list_size_total * sizeof(int), hipMemcpyHostToDevice);

    int* d_sizes;
    hipMalloc((void**)&d_sizes, size * sizeof(int));
    hipMemcpy(d_sizes, list_sizes, size * sizeof(int), hipMemcpyHostToDevice);

    int* changed = (int*)malloc(sizeof(int));
    int* d_changed;
    hipMalloc((void**)&d_changed, sizeof(int));
    *changed = 1;

    while (*changed != 0) {
        *changed = 0;
        hipMemcpy(d_changed, changed, sizeof(int), hipMemcpyHostToDevice);

        befriend_adjacents<<<1, size>>>(d_adj_lists, d_sizes, d_labels, d_changed);

        hipMemcpy(changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
    }

    hipMemcpy(labels, d_labels, size * sizeof(int), hipMemcpyDeviceToHost);

    print_labels(labels, size);

    hipFree(d_changed); hipFree(d_labels); hipFree(d_changed); hipFree(d_adj_lists);

    free(labels); free(changed);
}

int populate_array(vector<int>* arr, int* len) {
    ifstream infile( "inp.txt" );
    if (!infile.is_open()) {
        cout<<"File failed to open"<<endl;
        return 0;
    }
    string line;
    while (getline(infile, line))
    {
        istringstream ss(line);
        while (ss)
        {
            string s;
            if (!getline(ss, s, ',')) break;

            (*len)++;
            arr->push_back(atoi(s.c_str()));

        }
    }
    return 1;
}

int main() {
    // int * adj_lists[3];
    // int* list0;
    // int list1[1] = {2};
    // int list2[2] = {1}; 
    // adj_lists[0] = list0;
    // adj_lists[1] = list1;
    // adj_lists[2] = list2;

		/* Simple Test

    int adj_lists[4] = {1, 2, 0, 0};

    int size = 4;
    int sizes[4] = {2, 1, 1, 0};
		*/

		int adj_lists[20] = {1, 2, 0, 3, 0, 3, 1, 2, 5, 4, 7, 8, 6, 10, 6, 9, 10, 8, 7, 8};

    int size = 11;
    int sizes[11] = {2, 2, 2, 2, 1, 1, 2, 2, 3, 1, 2};

    print_lists(adj_lists, size, sizes);

    cc_para(adj_lists, size, sizes);
}
