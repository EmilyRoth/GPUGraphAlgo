#include <stdio.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <map>
#include <vector>

using namespace std;

class Edge {
	int src, dest, val;
}

class Graph {
	int V, E;
	Edge* edges;
}

class Segment {
	int rep;
	int rank;
	vector<Edge> adjList;
}

map<int, Segment*> segmentOwner;
vector<Segment*> segments;

void populateSegments(vector<Edge> edges) {
	for(vector<Edge>::iterator it = edges.begin(); it != edges.end(); ++it) {
		Segment* seg;
		if (segmentOwner.find(it->src) == segmentOwner.end()) {
			seg = new Segment();
		} else {
			seg = segmentOwner[it->src]
		}
		seg->adjList.push_back(*it);
		segmentOwner[it->src] = seg;
		if (segmentOwner.find(it->dest) == segmentOwner.end()) {
			seg = new Segment();
		} else {
			seg = segmentOwner[it->src]
		}
		seg->adjList.push_back(*it);
		segmentOwner[it->src] = seg;
	}
}

void boruvkaParallel(vector<Edge> edges) {
    int V = graph->V, E = graph->E;

    populateSegments(edges)

    int *cheapest = new int[]
}