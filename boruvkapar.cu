#include <stdio.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <map>
#include <vector>

using namespace std;

class Edge {
	int src, dest, val;
}

class Graph {
	int V, E;
	Edge* edges;
}

class Segment {
	int rep;
	int rank;
	vector<Edge> adjList;
}

map<int, Segment*> segmentOwner;
vector<Segment*> segments;

void populateSegments(vector<Edge> edges) {
	for(vector<Edge>::iterator it = edges.begin(); it != edges.end(); ++it) {
		Segment* seg;
		if (segmentOwner.find(it->src) == segmentOwner.end()) {
			seg = new Segment();
		} else {
			seg = segmentOwner[it->src]
		}
		seg->adjList.push_back(*it);
		segmentOwner[it->src] = seg;
		segments.push_back(seg);
		if (segmentOwner.find(it->dest) == segmentOwner.end()) {
			seg = new Segment();
		} else {
			seg = segmentOwner[it->src]
		}
		seg->adjList.push_back(*it);
		segmentOwner[it->src] = seg;
		segments.push_back(seg);
	}
}

void boruvkaParallel(vector<Edge> edges) {
    int V = graph->V, E = graph->E;

	populateSegments(edges)
	
	int numSegments = segments.size();

	int *cheapest = new int[numSegments];
	
	// TODO: Start loop
		// Iterate over segments (one thread each)
		// Add cheapest edge to mst
		// Set owner to the segment of the lower index member (atomic)
		// Merge adjacency lists
		// Get rid of old segments
		// Update numSegments
}